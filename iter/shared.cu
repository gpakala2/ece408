#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define SIZE 38

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    //printf("yikes");
    __shared__ float tile[SIZE][SIZE];

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    int b = blockIdx.z;

    int W_size = ceil(1.0 * Width_out / TILE_WIDTH);
    int m = blockIdx.x;
    int h = (blockIdx.y / W_size) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_size) * TILE_WIDTH + threadIdx.x;

    // int h_i = h - (K / 2);
    // int w_i = w - (K / 2);

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    float acc = 0.0f;

    for (int c = 0; c < Channel; c++) { // sum over all input channels
        if((h < Height) && (w < Width))
            tile[threadIdx.y][threadIdx.x] = in_4d(b, c, h, w);
        else
            tile[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();
        
        if(threadIdx.y < TILE_WIDTH && threadIdx.x < TILE_WIDTH){
            for (int i = 0; i < K; i++) // loop over KxK filter
                for (int j = 0; j < K; j++)
                    acc += tile[threadIdx.y + i][threadIdx.x + j] * mask_4d(m, c, i, j);
        }

        __syncthreads();
    }

    if (h < Height_out && w < Width_out && threadIdx.y < TILE_WIDTH && threadIdx.x < TILE_WIDTH) {
        out_4d(b, m, h, w) = acc;
    }


    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int outputSize = Batch * Map_out * Height_out * Width_out;
    int inputSize = Batch * Channel * Height * Width;
    int kernelSize = Map_out * Channel * K * K;

    printf("\nKernel Size: %d\n", kernelSize);
    printf("Channels: %d\n", Channel);
    printf("K: %d\n", K);

    // FILE *fp;
    // fp = fopen("out.txt", "w");

    // for (int i = 0; i < inputSize; i++) {
    //     fprintf(fp, "%.2f\n", host_input[i]);
    //     // check for error here too
    // }

    // fclose(fp);

    hipMalloc((void **) device_output_ptr, outputSize * sizeof(float));
    hipMalloc((void **) device_input_ptr, inputSize * sizeof(float));
    hipMalloc((void **) device_mask_ptr, kernelSize * sizeof(float));

    hipMemcpy(*device_input_ptr, host_input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, kernelSize * sizeof(float), hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;

    int Width_grid = ceil(1.0 * Width_out / TILE_WIDTH);    // Number of horizontal tiles for output maps
    int Height_grid = ceil(1.0 * Height_out / TILE_WIDTH);    // Numer of vertical tiles for output maps

    int Out_grid = Height_grid * Width_grid;   // Y-Dimension of the grid
    int INSIZE = TILE_WIDTH + K - 1;

    dim3 gridDim(Map_out, Out_grid, Batch);
    dim3 blockDim(INSIZE, INSIZE, 1);

    printf("\nMap out: %d\n", Map_out);
    printf("Out grid: %d\n", Out_grid);
    printf("Batch: %d\n", Batch);
    printf("Size: %d\n", INSIZE);

    conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    int Height_out = Height - K + 1;
    int Width_out = Width - K + 1;
    
    int outputSize = Batch * Map_out * Height_out * Width_out;

    hipMemcpy(host_output, device_output, outputSize * sizeof(float), hipMemcpyDeviceToHost);

    if(Channel == 1){
        FILE *fp;
        fp = fopen("ref.txt", "w");

        for (int i = 0; i < outputSize; i++) {
            fprintf(fp, "%.2f\n", host_output[i]);
            // check for error here too
        }
        fclose(fp);
    }

    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    // hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
